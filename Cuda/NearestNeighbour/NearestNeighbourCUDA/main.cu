#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <conio.h>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void FindClosestGPU(float3* points, int* indices, int count);

int main()
{
	srand(1);

	// Number of points
	const int count{ 100000 };

	// Array of points
	int* indexOfClosest = new int[count];
	float3* points = new float3[count];

	float3* d_points;
	int* d_indexOfClosest;

	// Create a list of random points
	for (int i = 0; i < count; i++)
	{
		points[i].x = (float)((rand() % 10000) - 5000);
		points[i].y = (float)((rand() % 10000) - 5000);
		points[i].z = (float)((rand() % 10000) - 5000);
	}

	hipMalloc(&d_points, sizeof(float3) * count);
	hipMemcpy(d_points, points, sizeof(float3) * count, hipMemcpyHostToDevice);
	hipMalloc(&d_indexOfClosest, sizeof(int) * count);

	// This variable is used to keep track of the fastest time so far
	long fastest{ 1000000 };

	// Run the algorithm 20 times
	for (int q = 0; q < 20; q++)
	{
		long startTime = clock();

		FindClosestGPU<<<(count / 320)+1, 320>>>(d_points, d_indexOfClosest, count);
		hipMemcpy(indexOfClosest, d_indexOfClosest, sizeof(int) * count, hipMemcpyDeviceToHost);

		long finishTime = clock();

		std::cout << "Run " << q << " took " << (finishTime - startTime) << "ms" << std::endl;

		// If that run was faster update the fastest time so far
		if ((finishTime - startTime) < fastest)
			fastest = (finishTime - startTime);
	}

	// Print out the fastest time
	std::cout << "Fastest time: " << fastest << std::endl;

	// Print the final results to screen
	std::cout << "Final results:\n";
	for (int i = 0; i < 10; i++)
		std::cout << i << "." << indexOfClosest[i] << std::endl;

	//Deallocate ram
	delete[] indexOfClosest;
	delete[] points;
	hipFree(d_points);
	hipFree(d_indexOfClosest);

	hipDeviceReset();

	return 0;
}