#include <hip/hip_runtime.h>



__global__ void FindClosestGPU(float3* points, int* indices, int count)
{
	if (count <= 1) return;

	auto square = [](const auto& num) { return num * num; };

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < count)
	{
		float3 thisPoint = points[idx];
		float smallestDistSoFar = 3.40282e38f;

		for (int i = 0; i < count; i++)
		{
			if (i == idx) continue;
			float dist = (
				square(thisPoint.x - points[i].x) +
				square(thisPoint.y - points[i].y) +
				square(thisPoint.z + points[i].z)
				);
			if (dist < smallestDistSoFar)
			{
				smallestDistSoFar = dist;
				indices[idx] = i;
			}
		}
	}
}