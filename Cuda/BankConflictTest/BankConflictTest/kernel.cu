#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <conio.h>

typedef unsigned long long uint_64;

struct Coord4D
{
	float x, y, z, w;
	float padding;
};

__global__ void MyKernel(uint_64* time)
{
	__shared__ Coord4D shared[512];
	uint_64 startTime = clock();

	shared[threadIdx.x].x++;

	uint_64 finishTime = clock();
	*time = finishTime - startTime;
}

int main()
{
	uint_64 time;
	uint_64* d_time;
	hipMalloc(&d_time, sizeof(uint_64));

	for (int i = 0; i < 10; i++)
	{
		MyKernel<<<1, 32>>>(d_time);
		hipMemcpy(&time, d_time, sizeof(uint_64), hipMemcpyDeviceToHost);

		// 14 is the overhead for calling clock
		std::cout << "Time: " << (time - 14) / 32 << "\n\n";
	}

	hipFree(d_time);

	_getch();
	hipDeviceReset();
	return 0;
}