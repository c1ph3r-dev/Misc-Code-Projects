﻿#include "Exponential-Cuda.cuh"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <ostream>
#include <chrono>

namespace TIMER {
	struct Timer
	{
		std::chrono::time_point<std::chrono::steady_clock> start, end;
		std::chrono::duration<float> duration;

		Timer()
		{
			Reset();
		}

		~Timer()
		{

		}

		inline void Reset() noexcept
		{
			start = std::chrono::high_resolution_clock::now();
		}

		void SetEnd() noexcept
		{
			end = std::chrono::high_resolution_clock::now();
			duration = end - start;
		}

		inline float GetTimeInMS() const noexcept
		{
			return float(duration.count() * 1000.f);
		}

		inline float GetTimeInS() const noexcept
		{
			return float(duration.count());
		}
	};
}

int main()
{
	using namespace JRAMPERSAD::EXPONENTIAL;

	std::vector<int64_t> vec = { 1, -6, 11, -6 };

	Function g(3);
	g.SetConstants(vec);
	ExponentialCUDA g_CUDA(3);
	g_CUDA.SetConstants(vec);

	GA_Options options;
	options.mutation_percentage = 0.005;
	options.num_of_generations = 25;
	options.sample_size = 1000;
	options.data_size = 5000000;
	options.min_range = -100;
	options.max_range = 100;

	TIMER::Timer t;
	auto res = g.get_real_roots(options);
	t.SetEnd();

	TIMER::Timer t_CUDA;
	res = g_CUDA.get_real_roots(options);
	t_CUDA.SetEnd();

	std::cout << "CPU took: " << t.GetTimeInMS() << "ms\n";
	std::cout << "GPU took: " << t_CUDA.GetTimeInMS() << "ms\n";
}