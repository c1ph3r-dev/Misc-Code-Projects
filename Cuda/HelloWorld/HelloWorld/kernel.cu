﻿#include "hip/hip_runtime.h"
//#include "device_launch_parameters.h"

#include <iostream>
#include <stdlib.h>
#include <ctime>

__global__ void AddInts(int* a, int* b, int count)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < count)
	{
		a[id] += b[id];
	}
}

int main()
{
	srand(time(NULL));
	int count = 256 * 256;
	int *h_a = new int[count];
	int *h_b = new int[count];

	for (int i = 0; i < count; i++)
	{
		h_a[i] = rand() % 1000;
		h_b[i] = rand() % 1000;
	}

	std::cout << "Prior to addition:\n";
	for (int i = 0; i < 5; i++)
		std::cout << h_a[i] << " " << h_b[i] << std::endl;

	int *d_a, *d_b;

	if (hipMalloc(&d_a, sizeof(int) * count) != hipSuccess)
	{
		delete[] h_a;
		delete[] h_b;
		std::cout << "Nope! No\n";
		return 0;
	}
	if (hipMalloc(&d_b, sizeof(int) * count) != hipSuccess)
	{
		std::cout << "Nope! No\n";
		hipFree(d_a);
		delete[] h_a;
		delete[] h_b;
		return 0;
	}

	if (hipMemcpy(d_a, h_a, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cout << "Could not copy!\n";
		hipFree(d_a);
		hipFree(d_b);
		delete[] h_a;
		delete[] h_b;
		return 0;
	}

	if (hipMemcpy(d_b, h_b, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cout << "Could not copy!\n";
		hipFree(d_a);
		hipFree(d_b);
		delete[] h_a;
		delete[] h_b;
		return 0;
	}

	AddInts<<<count / 256 + 1, 256>>>(d_a, d_b, count);

	if (hipMemcpy(h_a, d_a, sizeof(int) * count, hipMemcpyDeviceToHost) != hipSuccess)
	{
		hipFree(d_a);
		hipFree(d_b);
		delete[] h_a;
		delete[] h_b;
		std::cout << "Nope!\n";
		return 0;
	}

	std::cout << "After addition:\n";
	for (int i = 0; i < 5; i++)
		std::cout << "It's " << h_a[i] << std::endl;


	hipFree(d_a);
	hipFree(d_b);
	delete[] h_a;
	delete[] h_b;

	return 0;
}